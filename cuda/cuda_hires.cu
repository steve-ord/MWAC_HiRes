#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <complex.h>
#include "cuda_hires.h"

static __global__ void Unpack_4b(complex_sample_4b_t *, hipfftComplex *, int,int,int,int);
static __global__ void BlockReorderTranspose(hipfftComplex *, complex_sample_4b_t *);
void test_fft() {
#define NSAMP 8
    float bpass[NSAMP];

    // ramp bandpass
    for (int b = 0; b < NSAMP; b++) {
        bpass[b] = b - NSAMP/2.0;
    }
    // synthesis
    hipfftComplex x[NSAMP];
    for (int n=0 ; n < NSAMP ; n++) {
        x[n].x = 0.0;
        x[n].y = 0.0;
        for (int k=0 ; k < NSAMP ; k++) {
            float freq = k - NSAMP/2.0;
            x[n].x =  x[n].x + bpass[k]*cosf(2*M_PI*freq*n/NSAMP) ;
            x[n].y =  x[n].y + bpass[k]*sinf(2*M_PI*freq*n/NSAMP) ;
        }
    }

    // Allocate device memory
 
    hipfftComplex *d_x = NULL;
    checkCudaErrors(hipMalloc((void **) &d_x,NSAMP*sizeof(hipfftComplex)));

    // copy to device
    checkCudaErrors(hipMemcpy(d_x,x,NSAMP*sizeof(hipfftComplex),hipMemcpyHostToDevice));

    //FFT
    hipfftHandle plan; 
    checkCudaErrors(hipfftPlan1d(&plan, NSAMP, HIPFFT_C2C, 1)); 
    checkCudaErrors(hipfftExecC2C(plan, (hipfftComplex *)d_x, (hipfftComplex *)d_x, HIPFFT_FORWARD));

    // copy back
    checkCudaErrors(hipMemcpy(x,d_x,NSAMP*sizeof(hipfftComplex),hipMemcpyDeviceToHost));

    // check

    for (int i=0;i<NSAMP;i++) {
        
        x[i].x = x[i].x/NSAMP;
        x[i].y = x[i].y/NSAMP;

        float abs_x = x[i].x * x[i].x + x[i].y*x[i].y;
        fprintf(stdout,"%f --- %f\n",bpass[i],sqrtf(abs_x));
    }
}

void hires_4b(complex_sample_4b_t *in_data, complex_sample_4b_t *out_data, int chan_select, int nchan_in, int nchan_out, int ninputs) {
    
    // setDevice
    
    // Allocate device memory
    // input data is a 4 bit sample for ninputs * nchan_in * nchan_out (time_samples)
    // we need to pull out a nchan_out time_samples for channel==chan_select for each input
    // After the FFT we output a single time sample for an nchan_out FB for all the inputs
    // repacked into the correct order
    
    // is it worth stacking the FFT?

    int nsamples_in = ninputs * nchan_in * nchan_out;
    int nsamples_out = ninputs * nchan_out;

    // size of the FFT stack
    // ninput rows of nchan_out complex samples
    int nsamples_fft_stack = ninputs*nchan_out;

    complex_sample_4b_t *d_signal;
    checkCudaErrors(hipMalloc((void **) &d_signal,nsamples_in*1));
    
    complex_sample_t *d_fft_stack;
    checkCudaErrors(hipMalloc((void **) &d_fft_stack,nsamples_fft_stack*sizeof(complex_sample_t)));

    // copy data to Device
    checkCudaErrors(hipMemcpy(d_signal,in_data,nsamples_in,hipMemcpyHostToDevice));

    // call kernel to expand input data to float
    // assumes number of time samples == number of output channels
    // also extracts the time series for this channel and all inputs
    Unpack_4b<<<nchan_out,ninputs>>>(d_signal, d_fft_stack, chan_select,nchan_in,nchan_out,ninputs);
    getLastCudaError("Kernel execution failed [ Unpack_4b ]");

    // call kernel to batch FFT 
    // this will FFT all inputs at once
    hipfftHandle plan; 
    checkCudaErrors(hipfftPlan1d(&plan, nchan_out, HIPFFT_C2C, ninputs)); 
    checkCudaErrors(hipfftExecC2C(plan, (hipfftComplex *)d_fft_stack, (hipfftComplex *)d_fft_stack, HIPFFT_FORWARD));

    // call kernels to repack
    // We will perform the reordering and repacking in a single CUDA Kernel
    // We have to reorder the fft_stack which is now [input][channel] (channels moving fastest) to [channel][input]
    // and we need to reorder the channeld by block shifting the second half of the array to the beginning

    // probably best to do this in 2 stages:
   
    // channel reordering
 
    // & transpose & float to 4
    BlockReorderTranspose<<<ninputs,nchan_out>>>(d_fft_stack,d_signal);

    getLastCudaError("Kernel execution failed [ BlockReorderTranspose ]");

    // data back to host

    checkCudaErrors(hipMemcpy(out_data,d_signal,nsamples_out,hipMemcpyDeviceToHost));

    // free device memory
    checkCudaErrors(hipFree(d_signal));
    checkCudaErrors(hipFree(d_fft_stack));
}
static __global__ void BlockReorderTranspose(complex_sample_t *in, complex_sample_4b_t *out){

    const int input_location = blockIdx.x * gridDim.x + threadIdx.x;
    int output_location = 0;
    if (threadIdx.x < gridDim.x/2) {
        output_location = (threadIdx.x + gridDim.x/2)* blockDim.x + blockIdx.x;
    }
    else {
        output_location = (threadIdx.x - gridDim.x/2)* blockDim.x + blockIdx.x;
    }

    hipfftComplex sample = in[input_location];
    int8_t sample_x = __float2int_ru(sample.x); 
    int8_t sample_y = __float2int_ru(sample.y); 
    

    out[output_location] = 0x0;
    out[output_location] = (sample_y & 0xf);
    
    sample_x = sample_x & 0xf;

    out[output_location] = out[output_location] | (sample_x << 4);

}
static __global__ void Unpack_4b(complex_sample_4b_t *in_raw, complex_sample_t *fft_stack, int chan_select,int nchan_in,int nchan_out, int ninput){
    // we only need to unpack those entries corresponding to the 
    // channel we wish to process
    // the input data is in [time][freq][input] order - input moving fastest
    
    // this is called with ntime blocks of ninput threads

    int input_sample_offset = ninput*chan_select;
    const int input_id = threadIdx.x;
    const int time_id = blockIdx.x;

    uint64_t ii = (time_id*ninput*nchan_in) + input_id + input_sample_offset;

    // unpack this sample and put the result in 
    // in transposed FFT_STACK at:
    uint64_t out_location = (input_id*blockDim.x) + time_id;

    complex_sample_4b_t sample = in_raw[ii];
    
    // mask off the lowest 4
    
    uint8_t original = sample & 0xf;
    

    if (original >= 0x8) {
       fft_stack[out_location].y = original - 0x10;
    }
    else {
       fft_stack[out_location].y = original;
    }

    sample >>= 4;

    original = sample & 0xf;
    

    if (original >= 0x8) {
       fft_stack[out_location].x = original - 0x10;
    }
    else {
       fft_stack[out_location].x = original;
    }

}
